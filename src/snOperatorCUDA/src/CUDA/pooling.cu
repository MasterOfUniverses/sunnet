#include "hip/hip_runtime.h"
﻿//
// sunnet project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/sunnet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include "../stdafx.h"
#include "snOperatorCUDA/src/Operator/pooling.h"

using namespace std;
using namespace SN_Base;


struct gpuParams{

    hipdnnHandle_t cudnn = 0;
    hipdnnPoolingDescriptor_t pool_desc = 0;
    hipdnnTensorDescriptor_t in_desc = 0;
    hipdnnTensorDescriptor_t out_desc = 0;
    hipdnnTensorDescriptor_t grin_desc = 0;
    hipdnnTensorDescriptor_t grout_desc = 0;    
   
};

void Pooling::iniParamCUDA(bool isLern, const snSize& insz, const snSize& outsz, const poolParams& prms, void** pGpuPrm){
     
    bool isFirst = false;

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
    if (!gpuPrm){
  
        hipDeviceProp_t cu_deviceProps;
        hipGetDeviceProperties(&cu_deviceProps, 0);
        if (cu_deviceProps.major < 3){
            ERROR_MESS("%s requires SM >= 3.0");
            return;
        }
        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;
       
        hipdnnHandle_t cudnn = nullptr;
        cuCHECK(hipdnnCreate(&cudnn));
        gpuPrm->cudnn = cudnn;              

        isFirst = true;
    }

    // input
    hipdnnTensorDescriptor_t in_desc = nullptr;
    cuCHECK(hipdnnCreateTensorDescriptor(&in_desc));
    cuCHECK(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(insz.n), int(insz.d), int(insz.h), int(insz.w)));
    if (!isFirst)
        cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->in_desc));
    gpuPrm->in_desc = in_desc;
     
    // pool
    hipdnnPoolingDescriptor_t pool_desc = nullptr;
    cuCHECK(hipdnnCreatePoolingDescriptor(&pool_desc));

    hipdnnPoolingMode_t poolT = hipdnnPoolingMode_t::HIPDNN_POOLING_MAX;
    if (prms.type == poolType::avg)
        poolT = hipdnnPoolingMode_t::HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
   
    cuCHECK(hipdnnSetPooling2dDescriptor(pool_desc, poolT, hipdnnNanPropagation_t::HIPDNN_NOT_PROPAGATE_NAN,
        int(prms.kernel), int(prms.kernel), int(prms.paddingH), int(prms.paddingW), int(prms.stride), int(prms.stride)));
    if (!isFirst)
        cuCHECK(hipdnnDestroyPoolingDescriptor(gpuPrm->pool_desc));
    gpuPrm->pool_desc = pool_desc;

    // output
    int out_n = 0, out_c = 0, out_h = 0, out_w = 0;
    cuCHECK(hipdnnGetPooling2dForwardOutputDim(pool_desc, in_desc,
        &out_n, &out_c, &out_h, &out_w));

    if (outsz != snSize(out_w, out_h, out_c, out_n)){
        ERROR_MESS("CUDA error: outsz != snSize(out_w, out_h, out_c, out_n)");
        return;
    }

    hipdnnTensorDescriptor_t out_desc;
    cuCHECK(hipdnnCreateTensorDescriptor(&out_desc));
    cuCHECK(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));
    if (!isFirst)
        cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->out_desc));
    gpuPrm->out_desc = out_desc;

    if (isLern){
        // grout
        hipdnnTensorDescriptor_t grout_desc;
        cuCHECK(hipdnnCreateTensorDescriptor(&grout_desc));
        cuCHECK(hipdnnSetTensor4dDescriptor(grout_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(insz.n), int(insz.d), int(insz.h), int(insz.w)));
        if (!isFirst)
            cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->grout_desc));
        gpuPrm->grout_desc = grout_desc;

        // grin
        hipdnnTensorDescriptor_t grin_desc;
        cuCHECK(hipdnnCreateTensorDescriptor(&grin_desc));
        cuCHECK(hipdnnSetTensor4dDescriptor(grin_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            out_n, out_c, out_h, out_w));
        if (!isFirst)
            cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->grin_desc));
        gpuPrm->grin_desc = grin_desc;
    }

}

void Pooling::freeParamCUDA(void* gpuPrms){
    
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;

    if (!gpuPrm) return;
    
    cuCHECK(hipdnnDestroy(gpuPrm->cudnn));
    cuCHECK(hipdnnDestroyPoolingDescriptor(gpuPrm->pool_desc));
    cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->in_desc));
    cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->out_desc));
      
    if (gpuPrm->grin_desc){ // isLern
      cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->grin_desc));
      cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->grout_desc));
    }
}

__global__ void cuFiltrNegative(snSize outsz, snFloat* out){

    out += blockIdx.x * outsz.w * outsz.h + blockIdx.y * outsz.w * outsz.h * outsz.d;
       
    unsigned int i = threadIdx.x;
    while (i < (outsz.w * outsz.h)){

        if (out[i] < 0)
           out[i] = 0.0;

        i += blockDim.x;
    }    
}

void Pooling::forwardCUDA(const poolParams& poolPrms, const snSize& insz, const snFloat* input,
    const snSize& outsz, snFloat* output, void* gpuPrms){
    
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
      
    // run
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(hipdnnPoolingForward(gpuPrm->cudnn,
        gpuPrm->pool_desc,
        &alpha,
        gpuPrm->in_desc,
        input,
        &beta,
        gpuPrm->out_desc,
        output));
   
    // filtrNegative
    dim3 dimBlock(128);
    dim3 dimGrid(int(outsz.d), int(outsz.n));
         
    cuFiltrNegative << < dimGrid, dimBlock >> >(outsz, output);
  
}

void Pooling::backwardCUDA(const poolParams& poolPrms, const snSize& outsz, const snFloat* output, const snFloat* gradIn,
    const snSize& insz, const snFloat* input, snFloat* gradOut, void* gpuPrms){
       
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    
    // run
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(hipdnnPoolingBackward(gpuPrm->cudnn,
        gpuPrm->pool_desc,
        &alpha,
        gpuPrm->out_desc,
        output,
        gpuPrm->grin_desc,
        gradIn,
        gpuPrm->in_desc,
        input,
        &beta,
        gpuPrm->grout_desc,
        gradOut));
     
}
