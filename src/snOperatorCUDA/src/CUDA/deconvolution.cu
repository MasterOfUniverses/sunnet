#include "hip/hip_runtime.h"
﻿//
// sunnet project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/sunnet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include "../stdafx.h"
#include "snOperatorCUDA/src/Operator/deconvolution.h"

using namespace std;
using namespace SN_Base;


struct gpuParams{

    hipdnnHandle_t cudnn = 0;
    hipdnnConvolutionDescriptor_t conv_desc = 0;
    hipdnnTensorDescriptor_t in_desc = 0;
    hipdnnTensorDescriptor_t out_desc = 0;
    hipdnnTensorDescriptor_t grin_desc = 0;
    hipdnnTensorDescriptor_t grout_desc = 0;
    hipdnnFilterDescriptor_t w_desc = 0;
    hipdnnFilterDescriptor_t dw_desc = 0;
    hipdnnTensorDescriptor_t bias_desc = 0;

    hipdnnConvolutionFwdAlgo_t algoFwd;
    hipdnnConvolutionBwdDataAlgo_t algoBwdData;
    hipdnnConvolutionBwdFilterAlgo_t algoBwdW;

    size_t wsFwdSz = 0;
    size_t wsBwdDataSz = 0;
    size_t wsBwdWSz = 0;
    size_t inszMem = 0;
  
    void* d_wsFwd = 0;
    void* d_wsBwdData = 0;
    void* d_wsBwdW = 0;

};

void Deconvolution::iniParamCUDA(bool isLern, const snSize& insz, const snSize& outsz,
    const deconvParams& prms, void** pGpuPrm){
   
    bool isFirst = false;

    gpuParams* gpuPrm = (gpuParams*)*pGpuPrm;
    if (!gpuPrm){

        hipDeviceProp_t cu_deviceProps;

        hipGetDeviceProperties(&cu_deviceProps, 0);
        if (cu_deviceProps.major < 3){
            ERROR_MESS("%s requires SM >= 3.0");
            return;
        }
        gpuPrm = new gpuParams();
        memset(gpuPrm, 0, sizeof(gpuParams));
        *pGpuPrm = gpuPrm;

        hipdnnHandle_t cudnn = nullptr;
        cuCHECK(hipdnnCreate(&cudnn));
        gpuPrm->cudnn = cudnn;

        isFirst = true;
    }
    
    // input
    hipdnnTensorDescriptor_t in_desc = nullptr;
    cuCHECK(hipdnnCreateTensorDescriptor(&in_desc));
    cuCHECK(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(insz.n), int(insz.d), int(insz.h), int(insz.w)));
    if (!isFirst)
        cuCHECK(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->in_desc));
    gpuPrm->in_desc = in_desc;
      
    // w      
    hipdnnFilterDescriptor_t w_desc = nullptr;
    cuCHECK(hipdnnCreateFilterDescriptor(&w_desc));
    cuCHECK(hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        int(insz.d), int(outsz.d), int(prms.fHeight), int(prms.fWidth)));
    if (!isFirst)
        cuCHECK(hipdnnDestroyFilterDescriptor((hipdnnFilterDescriptor_t)gpuPrm->w_desc));
    gpuPrm->w_desc = w_desc;
     
    // conv
    hipdnnConvolutionDescriptor_t conv_desc = nullptr;
    cuCHECK(hipdnnCreateConvolutionDescriptor(&conv_desc));
    cuCHECK(hipdnnSetConvolution2dDescriptor(conv_desc, 0, 0, int(prms.stride), int(prms.stride), 1, 1,
        HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    if (!isFirst)
        cuCHECK(hipdnnDestroyConvolutionDescriptor((hipdnnConvolutionDescriptor_t)gpuPrm->conv_desc));
    gpuPrm->conv_desc = conv_desc;

    // output
    hipdnnTensorDescriptor_t out_desc;
    cuCHECK(hipdnnCreateTensorDescriptor(&out_desc));
    cuCHECK(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        int(outsz.n), int(outsz.d), int(outsz.h), int(outsz.w)));
    if (!isFirst)
        cuCHECK(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->out_desc));
    gpuPrm->out_desc = out_desc;

    // algorithm
    hipdnnConvolutionBwdDataAlgo_t algoBwdData;
    cuCHECK(hipdnnGetConvolutionBackwardDataAlgorithm(gpuPrm->cudnn, w_desc, in_desc, conv_desc, out_desc,
        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algoBwdData));
    gpuPrm->algoBwdData = algoBwdData;
  
    // workspace
    size_t wsBwdDataSz = 0;
    cuCHECK(hipdnnGetConvolutionBackwardDataWorkspaceSize(gpuPrm->cudnn, w_desc, in_desc, conv_desc, out_desc, algoBwdData, &wsBwdDataSz));
    gpuPrm->wsBwdDataSz = wsBwdDataSz;


    size_t wsFwdSz = 0, wsBwdWSz = 0;
    if (isLern){
        // grin
        hipdnnTensorDescriptor_t grin_desc;
        cuCHECK(hipdnnCreateTensorDescriptor(&grin_desc));
        cuCHECK(hipdnnSetTensor4dDescriptor(grin_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            int(outsz.n), int(outsz.d), int(outsz.h), int(outsz.w)));
        if (!isFirst)
            cuCHECK(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->grin_desc));
        gpuPrm->grin_desc = grin_desc;

        // grout
        hipdnnTensorDescriptor_t grout_desc;
        cuCHECK(hipdnnCreateTensorDescriptor(&grout_desc));
        cuCHECK(hipdnnSetTensor4dDescriptor(grout_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, int(insz.n), int(insz.d), int(insz.h), int(insz.w)));
        if (!isFirst)
            cuCHECK(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->grout_desc));
        gpuPrm->grout_desc = grout_desc;

        // dw     
        hipdnnFilterDescriptor_t dw_desc = nullptr;
        cuCHECK(hipdnnCreateFilterDescriptor(&dw_desc));
        cuCHECK(hipdnnSetFilter4dDescriptor(dw_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
            int(insz.d), int(outsz.d), int(prms.fHeight), int(prms.fWidth)));
        if (!isFirst)
            cuCHECK(hipdnnDestroyFilterDescriptor((hipdnnFilterDescriptor_t)gpuPrm->dw_desc));
        gpuPrm->dw_desc = dw_desc;

        // bias
        hipdnnTensorDescriptor_t bias_desc;
        cuCHECK(hipdnnCreateTensorDescriptor(&bias_desc));
        cuCHECK(hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
            1, int(insz.d), 1, 1));
        if (!isFirst)
            cuCHECK(hipdnnDestroyTensorDescriptor((hipdnnTensorDescriptor_t)gpuPrm->bias_desc));
        gpuPrm->bias_desc = bias_desc;

        // algorithm
        hipdnnConvolutionFwdAlgo_t algoFwd;
        cuCHECK(hipdnnGetConvolutionForwardAlgorithm(gpuPrm->cudnn, grin_desc, w_desc, conv_desc, grout_desc,
            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algoFwd));
        gpuPrm->algoFwd = algoFwd;

        hipdnnConvolutionBwdFilterAlgo_t algoBwdW;
        cuCHECK(hipdnnGetConvolutionBackwardFilterAlgorithm(gpuPrm->cudnn, grin_desc, in_desc, conv_desc, dw_desc,
            HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algoBwdW));
        gpuPrm->algoBwdW = algoBwdW;

        // workspace       
        cuCHECK(hipdnnGetConvolutionForwardWorkspaceSize(gpuPrm->cudnn, grin_desc, w_desc, conv_desc, grout_desc, algoFwd, &wsFwdSz));
        gpuPrm->wsFwdSz = wsFwdSz;
                
        cuCHECK(hipdnnGetConvolutionBackwardFilterWorkspaceSize(gpuPrm->cudnn, grin_desc, in_desc, conv_desc, dw_desc, algoBwdW, &wsBwdWSz));
        gpuPrm->wsBwdWSz = wsBwdWSz;
    }

    if (isFirst){     
        cuCHECK(hipMalloc(&gpuPrm->d_wsBwdData, wsBwdDataSz));

        if (isLern){          
            cuCHECK(hipMalloc(&gpuPrm->d_wsFwd, wsFwdSz));
            cuCHECK(hipMalloc(&gpuPrm->d_wsBwdW, wsBwdWSz));
        }
    }
    else if (gpuPrm->inszMem < insz.size()){      

        cuCHECK(hipFree(gpuPrm->d_wsBwdData)); gpuPrm->d_wsBwdData = 0;  
        
        cuCHECK(hipMalloc(&gpuPrm->d_wsBwdData, wsBwdDataSz));

        if (isLern){
          
            cuCHECK(hipFree(gpuPrm->d_wsFwd));     gpuPrm->d_wsFwd = 0;
            cuCHECK(hipFree(gpuPrm->d_wsBwdW));    gpuPrm->d_wsBwdW = 0;           
        
            cuCHECK(hipMalloc(&gpuPrm->d_wsFwd, wsFwdSz));
            cuCHECK(hipMalloc(&gpuPrm->d_wsBwdW, wsBwdWSz));           
        }
        gpuPrm->inszMem = insz.size();
    }
}

void Deconvolution::freeParamCUDA(void* gpuPrms){
       
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;

    if (!gpuPrm) return;

    cuCHECK(hipdnnDestroy(gpuPrm->cudnn));
    cuCHECK(hipdnnDestroyConvolutionDescriptor(gpuPrm->conv_desc));
    cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->in_desc));
    cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->out_desc));   
    cuCHECK(hipdnnDestroyFilterDescriptor(gpuPrm->w_desc));        
   
    cuCHECK(hipFree(gpuPrm->d_wsBwdData));

    if (gpuPrm->grin_desc){ // isLern
        cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->grin_desc));
        cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->grout_desc));
        cuCHECK(hipdnnDestroyFilterDescriptor(gpuPrm->dw_desc));
        cuCHECK(hipdnnDestroyTensorDescriptor(gpuPrm->bias_desc));
      
        cuCHECK(hipFree(gpuPrm->d_wsFwd));
        cuCHECK(hipFree(gpuPrm->d_wsBwdW));
    }
}

void Deconvolution::forwardCUDA(const deconvParams& prms,
    const snFloat* weight, const snSize& insz, const snFloat* input, const snSize& outsz, snFloat* output, void* gpuPrms){
     
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
       
    // run
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(hipdnnConvolutionBackwardData(gpuPrm->cudnn,
        &alpha,
        gpuPrm->w_desc,
        weight,
        gpuPrm->in_desc,
        input,
        gpuPrm->conv_desc,
        gpuPrm->algoBwdData,
        gpuPrm->d_wsBwdData,
        gpuPrm->wsBwdDataSz,
        &beta,
        gpuPrm->out_desc,
        output));
   
}

__global__ void cuBwdBias(snSize insz, const snFloat* bias, snFloat* grout){

    size_t isz = insz.w * insz.h;

    snFloat* pGrOut = grout + isz * blockIdx.x + isz * insz.d * blockIdx.y;

    snFloat b = bias[blockIdx.x];

    unsigned int i = threadIdx.x;
    while (i < isz){
        
        pGrOut[i] += b;

        i += blockDim.x;
    }
}

void Deconvolution::backwardCUDA_GW(const deconvParams& prms,
    const snFloat* weight, const snSize& insz, const snFloat* input, const snSize& outsz, const snFloat* gradIn, snFloat* gradOut, snFloat* dWeightOut, void* gpuPrms){
 
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t wStepByN = prms.fWidth * prms.fHeight * insz.d * outsz.d;

    // run       
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(hipdnnConvolutionForward(gpuPrm->cudnn,
        &alpha,
        gpuPrm->grin_desc,
        gradIn,
        gpuPrm->w_desc,
        weight,
        gpuPrm->conv_desc,
        gpuPrm->algoFwd,
        gpuPrm->d_wsFwd,
        gpuPrm->wsFwdSz,
        &beta,
        gpuPrm->grout_desc,
        gradOut));

    cuCHECK(hipdnnConvolutionBackwardFilter(gpuPrm->cudnn,
        &alpha,        
        gpuPrm->grin_desc,
        gradIn,
        gpuPrm->in_desc,
        input,
        gpuPrm->conv_desc,
        gpuPrm->algoBwdW,
        gpuPrm->d_wsBwdW,
        gpuPrm->wsBwdWSz,
        &beta,
        gpuPrm->dw_desc,
        dWeightOut));

    cuCHECK(hipdnnConvolutionBackwardBias(gpuPrm->cudnn,
        &alpha,
        gpuPrm->in_desc,
        input,
        &beta,
        gpuPrm->bias_desc,
        dWeightOut + wStepByN));

    // +bias
    dim3 dimBlock(128);
    dim3 dimGrid(int(insz.d), int(insz.n));

    cuBwdBias << < dimGrid, dimBlock >> > (insz, weight + wStepByN, gradOut);
     
}

void Deconvolution::backwardCUDA_G(const deconvParams& prms,
    const snFloat* weight, const snSize& insz, const snSize& outsz, const snFloat* gradIn, snFloat* gradOut, void* gpuPrms){
    
    gpuParams* gpuPrm = (gpuParams*)gpuPrms;
    size_t wStepByN = prms.fWidth * prms.fHeight * insz.d * outsz.d;

    // run      
    snFloat alpha = 1.f, beta = 0.f;
    cuCHECK(hipdnnConvolutionForward(gpuPrm->cudnn,
        &alpha,
        gpuPrm->grin_desc,
        gradIn,
        gpuPrm->w_desc,
        weight,
        gpuPrm->conv_desc,
        gpuPrm->algoFwd,
        gpuPrm->d_wsFwd,
        gpuPrm->wsFwdSz,
        &beta,
        gpuPrm->grout_desc,
        gradOut));

    // +bias
    cuBwdBias << < int(insz.n), 128 >> > (insz, weight + wStepByN, gradOut);
       
}
