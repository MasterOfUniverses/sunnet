#include "hip/hip_runtime.h"
//
// sunnet project
// Copyright (C) 2018 by Contributors <https://github.com/Tyill/sunnet>
//
// This code is licensed under the MIT License.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files(the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and / or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.
//
#include "../stdafx.h"
#include "snOperatorCUDA/src/Operator/lossFunction.h"

using namespace std;
using namespace SN_Base;


__global__ void softMaxACrossEntropyFwd(snSize iosz, snFloat* inout){
      
    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
 
    inout += blockIdx.x * inStepByN;
           
    __shared__ int tmax;
    __shared__ snFloat tsumm;

    tmax = 0;
    tsumm = 0;

    __syncthreads();

    unsigned int i = threadIdx.x;
    while (i < inStepByN){

        atomicMax(&tmax, int(inout[i] * 100.F));  // TODO redo to reduction
       
        i += blockDim.x;
    }
   
    __syncthreads();

    i = threadIdx.x;
    while (i < inStepByN){
       
        inout[i] = ((inout[i] - tmax / 100.F) > -20) ? exp(inout[i] - tmax / 100.F) : 0.1E-8F;
                 
        atomicAdd(&tsumm, inout[i]); // TODO redo to reduction

        i += blockDim.x;
    }
  
    __syncthreads();
   
    i = threadIdx.x;
    while (i < inStepByN){

        inout[i] /= tsumm;

        i += blockDim.x;
    }   
}

__global__ void softMaxACrossEntropyBwd(snSize iosz, snFloat* out, snFloat* targ, snFloat* grad){

    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size  
    
    grad += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    out += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    targ += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int i = threadIdx.x;

    while (i < inStepByD){

        grad[i] = out[i] - targ[i];

        i += blockDim.x;
    } 
}

__global__ void binaryCrossEntropyBwd(snSize iosz, snFloat* out, snFloat* targ, snFloat* grad){

    size_t inStepByD = iosz.w * iosz.h,     // step out by input
           inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size  

    grad += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    out += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    targ += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int i = threadIdx.x;

    while (i < inStepByD){
        
        grad[i] = (out[i] - targ[i]) / (out[i] * (1.F - out[i]));

        i += blockDim.x;
    }
}

__global__ void regressionMSEBwd(snSize iosz, snFloat* out, snFloat* targ, snFloat* grad){

    size_t inStepByD = iosz.w * iosz.h,     // step out by input
        inStepByN = inStepByD * iosz.d;  // step out by batch       

    // gridDim.x - number of out layers
    // gridDim.y - batch size  

    grad += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    out += blockIdx.x * inStepByD + blockIdx.y * inStepByN;
    targ += blockIdx.x * inStepByD + blockIdx.y * inStepByN;

    unsigned int i = threadIdx.x;

    while (i < inStepByD){
        
        grad[i] = 2 * (out[i] - targ[i]) / inStepByN;

        i += blockDim.x;
    }
}


void lossForward(const snSize& sz, snFloat* inout, lossType loss){

    dim3 dimBlock(256);
    dim3 dimGrid(int(sz.n));

    switch (loss){
        case lossType::softMaxACrossEntropy:
            softMaxACrossEntropyFwd <<<dimGrid, dimBlock >>>(sz, inout);
            break;

        case lossType::binaryCrossEntropy:
            break;

        case lossType::regressionMSE: 
            break;
    }
}

void lossBackward(const snSize& sz, snFloat* out, snFloat* targ, snFloat* grad, lossType loss){

    dim3 dimBlock(128);
    dim3 dimGrid(int(sz.d), int(sz.n));

    switch (loss){
      case lossType::softMaxACrossEntropy:
          
          softMaxACrossEntropyBwd << <dimGrid, dimBlock >> >(sz, out, targ, grad); 
          break;    
      
      case lossType::binaryCrossEntropy:
      
          binaryCrossEntropyBwd << <dimGrid, dimBlock >> >(sz, out, targ, grad);
          break;
                                           
      case lossType::regressionMSE: // Mean Square Error
      
          regressionMSEBwd << <dimGrid, dimBlock >> >(sz, out, targ, grad);
          break;
    }   
}
